#include "hip/hip_runtime.h"
#include "KNN_GPU.cuh"

#include <cstdio>
#include <vector>
#include <hip/hip_runtime.h>

#include "cuda_header.cuh"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


// a struct for one entry in the training data
struct RatingTriple {
    int m;
    int u;
    int r;
};

// a struct for holding integer ranges, used in pointer arithmetic
struct Range {
    int start;
    int end;
};

__global__
void cuda_euclidean_kernel(const unsigned int *gpu_movie_data, const int n_movies,
                           const int n_features, float *gpu_movie_distances) {

    // compute the current thread index
    int thread_index = blockIdx.x * blockDim.x + threadIdx.x;

    // while the thread corresponds to a valid entry in the distance matrix
    while (thread_index < n_movies * n_movies) {

        // compute indices of movie pair
        int i = thread_index / n_movies;
        int j = thread_index % n_movies;

        if (i == j) {
            // movie is distance -1.0 from itself
            gpu_movie_distances[n_movies * i + j] = -1.0;

        }
        else {

            // calculate distance
            float dist = 0.0;
            for(int k = 0; k < n_features; k++) {

                float x_ik = (float) gpu_movie_data[i * n_features + k];
                float x_jk = (float) gpu_movie_data[j * n_features + k];

                dist += pow(x_ik - x_jk, 2);
            }

            gpu_movie_distances[n_movies * i + j] = dist; // pow(dist, 0.5);

        }

        // advance thread index
        thread_index += blockDim.x * gridDim.x;
    }

}


__global__
void cuda_prediction_kernel(const float *gpu_movie_distances,
                            const Range *gpu_user_ranges,
                            RatingTriple *gpu_user_ratings,
                            const int k,
                            const int n_movies,
                            const int pred_movies,
                            const int pred_users,
                            float *gpu_knn_predictions) {
                                
    // compute the current thread index
    int thread_index = blockIdx.x * blockDim.x + threadIdx.x;

    // while the thread corresponds to a valid query pair
    while (thread_index < pred_movies * pred_users) {

        // compute indices of movie, user pair
        int u = thread_index / pred_movies;
        int m = thread_index % pred_movies;
        
        int start = gpu_user_ranges[u].start;
        int end   = gpu_user_ranges[u].end;
        
        // if there are insufficient movies rated by this user, use default
        if (end - start < k) {
            gpu_knn_predictions[thread_index] = 0.0;
        }
        // otherwise, computed weighted average of top k movies rated by this user
        else {
            
            float *topk_weights = new float[k];
            
            // copy array before sorting in place
            RatingTriple* topk = new RatingTriple[end - start];
            for (int i = 0; i < end - start; i++) {
                topk[i] = gpu_user_ratings[start + i];
            }
            
            // find top k, using selection sort (in place)
            for (int i = 0; i < k; i++) {
                
                // we assume all coefficients are between -1 and 1, and larger = more similar
                int j_max = i;
                float max_coef = -1.0;
                
                // identify largest unsorted element
                for (int j = i; j < end - start; j++) {
                    
                    int m2 = topk[j].m;
                    float coef = gpu_movie_distances[m * n_movies + m2];
                    if (coef > max_coef) {
                        max_coef = coef;
                        j_max = j;
                    }
                    
                }
                
                // swap largest unsorted element with ith element
                RatingTriple tmp = topk[i];
                topk[i] = topk[j_max];
                topk[j_max] = tmp;
                
                topk_weights[i] = max_coef;
            }
            
            // computed weighted average of top k
            float weight_sum = 0.0;
            float rating_sum = 0.0;
            for (int i = 0; i < k; i++)
            {
                rating_sum += topk_weights[i] * ((float) topk[i].r);
                weight_sum += topk_weights[i];
            }
            
            gpu_knn_predictions[thread_index] = 
                (weight_sum == 0.0) ? 0.0 : rating_sum / weight_sum;
            
            delete[] topk_weights;
            delete[] topk;
        }

        // advance thread index
        thread_index += blockDim.x * gridDim.x;
    }
    
}

int cuda_call_euclidean_kernel(const unsigned int blocks,
                               const unsigned int threads_per_block,
                               const unsigned int *movie_data,
                               const int n_movies,
                               const int n_features,
                               float *movie_distances) {

    // allocate and copy data to gpu memory
    unsigned int* gpu_movie_data;
    gpuErrchk( hipMalloc((void **) &gpu_movie_data, n_movies * n_features * sizeof(unsigned int)) );
    hipMemcpy(gpu_movie_data, movie_data, n_movies * n_features * sizeof(unsigned int), hipMemcpyHostToDevice);

    float* gpu_movie_distances;
    gpuErrchk( hipMalloc((void **) &gpu_movie_distances, n_movies * n_movies * sizeof(float)) );

    // call kernel
    cuda_euclidean_kernel<<<blocks, threads_per_block>>>(gpu_movie_data, n_movies, n_features, gpu_movie_distances);

    // check for errors on kernel call
    hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "Error %s\n", hipGetErrorString(err));
        return 0;
    }
    else {
        fprintf(stderr, "No kernel error detected\n");
    }

    // copy output to cpu memory
    hipMemcpy(movie_distances, gpu_movie_distances, n_movies * n_movies * sizeof(float), hipMemcpyDeviceToHost);

    // free gpu memory
    hipFree(gpu_movie_data);
    hipFree(gpu_movie_distances);

    return 1; // return success flag (for now)
}


int cuda_call_prediction_kernel(const unsigned int blocks,
                                const unsigned int threads_per_block,
                                std::vector<std::unordered_map<int, int>> &user_ratings,
                                const float *movie_distances, 
                                const int k,
                                const int n_movies,
                                const int n_users,
                                float *knn_predictions,
                                const int pred_movies,
                                const int pred_users) {
                                    
    // convert distances to similarity coefficients
    float * movie_distances_2 = new float [n_movies * n_movies];
    for (int i = 0; i < n_movies * n_movies; i++) {
        movie_distances_2[i] = 1.0 / (1.0 + movie_distances[i]);
    }
    
    // convert unordered map to array to load onto GPU
    
    // first, calculate total number of entries
    int n_ratings = 0;
    for (int u = 0; u < n_users; u++)
        n_ratings += user_ratings[u].size();
    
    // all ratings, grouped by user index
    RatingTriple* user_ratings_arr = new RatingTriple[n_ratings];
    // range of indices in user_ratings_arr corresponding to each user
    Range* user_ranges = new Range[n_users];
    int i = 0;
    
    for (int u = 0; u < n_users; u++) {
        
        user_ranges[u].start = i;
        
        // loop through unordered map for this user
        for (auto mr: user_ratings[u]) {
            
            int m = mr.first;
            int r = mr.second;
            
            user_ratings_arr[i] = {m, u, r};
            i += 1;
        }
        
        user_ranges[u].end = i;
    }
    
    // allocate and copy data to gpu memory
    float* gpu_movie_distances;
    gpuErrchk( hipMalloc((void **) &gpu_movie_distances, n_movies * n_movies * sizeof(float)) );
    hipMemcpy(gpu_movie_distances, movie_distances_2, n_movies * n_movies * sizeof(float), hipMemcpyHostToDevice);
    
    RatingTriple* gpu_user_ratings;
    gpuErrchk( hipMalloc((void **) &gpu_user_ratings, n_ratings * sizeof(RatingTriple)) );
    hipMemcpy(gpu_user_ratings, user_ratings_arr, n_ratings * sizeof(RatingTriple), hipMemcpyHostToDevice);
    
    Range* gpu_user_ranges;
    gpuErrchk( hipMalloc((void **) &gpu_user_ranges, n_users * sizeof(Range)) );
    hipMemcpy(gpu_user_ranges, user_ranges, n_users * sizeof(Range), hipMemcpyHostToDevice);
    
    float* gpu_knn_predictions;
    gpuErrchk( hipMalloc((void **) &gpu_knn_predictions, pred_movies * pred_users * sizeof(float)) );
    
    // call kernel
    cuda_prediction_kernel<<<blocks, threads_per_block>>>(gpu_movie_distances,
                                                          gpu_user_ranges,
                                                          gpu_user_ratings,
                                                          k,
                                                          n_movies,
                                                          pred_movies,
                                                          pred_users,
                                                          gpu_knn_predictions);
    
    // check for errors on kernel call
    hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "Error %s\n", hipGetErrorString(err));
        return 0;
    }
    
    // copy output to cpu memory
    hipMemcpy(knn_predictions, gpu_knn_predictions, pred_movies * pred_users * sizeof(float), hipMemcpyDeviceToHost);
    
    // free cpu memory
    delete[] user_ratings_arr;
    delete[] user_ranges;
    delete[] movie_distances_2;
    
    // free gpu memory
    hipFree(gpu_user_ratings);
    hipFree(gpu_user_ranges);
    hipFree(gpu_knn_predictions);

    return 1; // return success flag (for now)
}


